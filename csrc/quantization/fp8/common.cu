#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/all.h>
#include <c10/cuda/CUDAGuard.h>

#include <cmath>

#include "cuda_compat.h"
#include "dispatch_utils.h"

#ifdef USE_ROCM
  #include "amd/quant_utils.cuh"
#else
  #include "nvidia/quant_utils.cuh"
#endif

namespace vllm {

__device__ __forceinline__ float atomicMaxFloat(float* addr, float value) {
  float old;
  old = (value >= 0)
            ? __int_as_float(atomicMax((int*)addr, __float_as_int(value)))
            : __uint_as_float(
                  atomicMin((unsigned int*)addr, __float_as_uint(value)));

  return old;
}

#define FP8_E4M3_MAX std::numeric_limits<c10::Float8_e4m3fn>::max()

template <typename scalar_t>
__device__ __forceinline__ c10::Float8_e4m3fn scaled_fp8_conversion(
    const scalar_t val, const float inverted_scale) {
  float x = static_cast<float>(val) * inverted_scale;
  float r = fmax(-FP8_E4M3_MAX, fmin(x, FP8_E4M3_MAX));
  return static_cast<c10::Float8_e4m3fn>(r);
}

// Compute the absolute maximum m of the input tensor and store
// m / float8_e4m3::max() in *scale. Each thread block performs a
// reduction tree and the memory in scale is atomically updated.
// So to get the right answer, *scale needs to be initialized to
// a value <= 0.0 and we need to wait for all thread blocks to
// finish before consuming *scale.
template <typename scalar_t>
__global__ void segmented_max_reduction(float* __restrict__ scale,
                                        const scalar_t* __restrict__ input,
                                        int64_t num_elems) {
  __shared__ float cache[1024];
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  // First store maximum for all values processes by
  // the current thread in cache[threadIdx.x]
  scalar_t tmp = 0.0;
  while (i < num_elems) {
    float x = static_cast<float>(input[i]);
    tmp = max(tmp, fabs(x));
    i += blockDim.x * gridDim.x;
  }
  cache[threadIdx.x] = tmp;

  __syncthreads();

  // Now perform parallel reduction within the thread block
  int ib = blockDim.x / 2;
  while (ib != 0) {
    if (threadIdx.x < ib && cache[threadIdx.x + ib] > cache[threadIdx.x]) {
      cache[threadIdx.x] = cache[threadIdx.x + ib];
    }
    __syncthreads();
    ib /= 2;
  }
  // Finally, since cache[0] contains the maximum for this thread block,
  // atomically write the max to the target location
  if (threadIdx.x == 0) {
    atomicMaxFloat(scale,
                   cache[0] / std::numeric_limits<c10::Float8_e4m3fn>::max());
  }
}

template <typename scalar_t>
struct __align__(8) vec4_t {
  scalar_t x;
  scalar_t y;
  scalar_t z;
  scalar_t w;
};

typedef struct __align__(4) {
  c10::Float8_e4m3fn x;
  c10::Float8_e4m3fn y;
  c10::Float8_e4m3fn z;
  c10::Float8_e4m3fn w;
}
float8x4_t;

template <typename scalar_t>
__global__ void scaled_fp8_quant_kernel(c10::Float8_e4m3fn* __restrict__ out,
                                        const scalar_t* __restrict__ input,
                                        const float* __restrict__ scale,
                                        int64_t num_elems) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;

  // Invert the scale so that we can use multiplications to avoid expensive
  // division.
  const float inverted_scale = 1.0f / (*scale);

  // Vectorized input/output to better utilize memory bandwidth.
  const vec4_t<scalar_t>* vectorized_in =
      reinterpret_cast<const vec4_t<scalar_t>*>(input);
  float8x4_t* vectorized_out = reinterpret_cast<float8x4_t*>(out);

  int num_vec_elems = num_elems >> 2;

#pragma unroll 4
  for (int i = tid; i < num_vec_elems; i += blockDim.x * gridDim.x) {
    vec4_t<scalar_t> in_vec = vectorized_in[i];
    float8x4_t out_vec;

    out_vec.x = scaled_fp8_conversion(in_vec.x, inverted_scale);
    out_vec.y = scaled_fp8_conversion(in_vec.y, inverted_scale);
    out_vec.z = scaled_fp8_conversion(in_vec.z, inverted_scale);
    out_vec.w = scaled_fp8_conversion(in_vec.w, inverted_scale);
    vectorized_out[i] = out_vec;
  }

  // Handle the remaining elements if num_elems is not divisible by 4
  for (int i = num_vec_elems * 4 + tid; i < num_elems;
       i += blockDim.x * gridDim.x) {
    out[i] = scaled_fp8_conversion(input[i], inverted_scale);
  }
}

template <typename Tout, typename Tin, int Vec_size>
__global__ void convert_fp8_kernel(const Tin* __restrict__ src_data,
                                   Tout* __restrict__ dst_data,
                                   const float* scale, size_t N) {
  using V_in_vec = typename Vec<Tin, Vec_size>::Type;
  using V_out_vec = typename Vec<Tout, Vec_size>::Type;
  auto dst_data_vec = reinterpret_cast<V_out_vec*>(dst_data);
  auto src_data_vec = reinterpret_cast<const V_in_vec*>(src_data);

  int64_t startIdx = (threadIdx.x + blockDim.x * blockIdx.x);
  auto idx = startIdx;
  if (idx >= N) {
    return;
  }
#if defined(ENABLE_FP8)
  #ifdef USE_ROCM
  dst_data_vec[idx] = fp8::scaled_vec_conversion<V_out_vec, V_in_vec>(
      src_data_vec[idx], *scale);
  #else
  dst_data_vec[idx] =
      fp8::scaled_convert<V_out_vec, V_in_vec, Fp8KVCacheDataType::kFp8E4M3>(
          src_data_vec[idx], *scale);
  #endif
#else
  assert(false);
#endif
}

}  // namespace vllm

void static_scaled_fp8_quant(torch::Tensor& out,    // [..., d]
                             torch::Tensor& input,  // [..., d]
                             torch::Tensor& scale)  // [1]
{
  int64_t num_tokens = input.numel() / input.size(-1);
  int64_t num_elems = input.numel();
  dim3 grid(num_tokens);
  dim3 block(1024);
  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(
      input.scalar_type(), "scaled_fp8_quant_kernel", [&] {
        vllm::scaled_fp8_quant_kernel<scalar_t><<<grid, block, 0, stream>>>(
            out.data_ptr<c10::Float8_e4m3fn>(), input.data_ptr<scalar_t>(),
            scale.data_ptr<float>(), num_elems);
      });
}

void dynamic_scaled_fp8_quant(torch::Tensor& out,    // [..., d]
                              torch::Tensor& input,  // [..., d]
                              torch::Tensor& scale)  // [1]
{
  int64_t num_tokens = input.numel() / input.size(-1);
  int64_t num_elems = input.numel();
  dim3 grid(num_tokens);
  dim3 block(1024);
  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(
      input.scalar_type(), "scaled_fp8_quant_kernel", [&] {
        vllm::segmented_max_reduction<scalar_t><<<grid, block, 0, stream>>>(
            scale.data_ptr<float>(), input.data_ptr<scalar_t>(), num_elems);
        vllm::scaled_fp8_quant_kernel<scalar_t><<<grid, block, 0, stream>>>(
            out.data_ptr<c10::Float8_e4m3fn>(), input.data_ptr<scalar_t>(),
            scale.data_ptr<float>(), num_elems);
      });
}

template <typename Tout, typename Tin, int Vec_size>
struct call_convert_fp8 {
  void operator()(torch::Tensor const& src_data, torch::Tensor& dst_data,
                  torch::Tensor const& scale) {
    const auto N = src_data.numel() / Vec_size;
#ifdef USE_ROCM
    constexpr dim3 numThreads{1024, 1, 1};
#else
    constexpr dim3 numThreads{512, 1, 1};
#endif
    uint32_t numBlocks = (N + numThreads.x - 1) / numThreads.x;
    const dim3 grid{numBlocks, 1, 1};
    const auto stream = at::cuda::getCurrentCUDAStream();

    vllm::convert_fp8_kernel<Tout, Tin, Vec_size>
        <<<grid, numThreads, 0, stream>>>(
            reinterpret_cast<Tin*>(src_data.data_ptr()),
            reinterpret_cast<Tout*>(dst_data.data_ptr()),
            (float*)scale.data_ptr(), N);
  }
};

void convert_fp8(torch::Tensor& dst_data, torch::Tensor const& src_data,
                 torch::Tensor const& scale) {
  torch::Device src_device = src_data.device();
  torch::Device dst_device = dst_data.device();
  TORCH_CHECK(src_device.is_cuda(), "src must be on a GPU")
  TORCH_CHECK(dst_device.is_cuda(), "dst must be on a GPU")
  TORCH_CHECK(src_device.index() == dst_device.index(),
              "src and dst must be on the same GPU");
  at::cuda::OptionalCUDAGuard device_guard(src_device);
  auto t1 = src_data.dtype();
  auto t2 = dst_data.dtype();
#ifdef USE_ROCM
  constexpr int VecSize = 2;
#else
  constexpr int VecSize = 1;
#endif
  if (src_data.dtype() == at::ScalarType::Float) {
    call_convert_fp8<uint8_t, float, VecSize>{}(src_data, dst_data, scale);
  } else if (src_data.dtype() == at::ScalarType::Half) {
    call_convert_fp8<uint8_t, uint16_t, VecSize>{}(src_data, dst_data, scale);
  } else if (src_data.dtype() == at::ScalarType::BFloat16) {
    call_convert_fp8<uint8_t, __hip_bfloat16, VecSize>{}(src_data, dst_data,
                                                        scale);
  } else if (dst_data.dtype() == at::ScalarType::Float) {
    call_convert_fp8<float, uint8_t, VecSize>{}(src_data, dst_data, scale);
  } else if (dst_data.dtype() == at::ScalarType::Half) {
    call_convert_fp8<uint16_t, uint8_t, VecSize>{}(src_data, dst_data, scale);
  } else if (dst_data.dtype() == at::ScalarType::BFloat16) {
    call_convert_fp8<__hip_bfloat16, uint8_t, VecSize>{}(src_data, dst_data,
                                                        scale);
  }
}